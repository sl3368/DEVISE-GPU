#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdlib.h> 
 
// error checking for CUDA calls: use this around ALL your calls!
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
 
 
//need multiple kernels for different types of dot products
//and outer products mainly

 
// same as above, only for GPU: cannot return values, so must store
// result in global memory location ("count")
// also: must make sure this thread maps to useful data! (what if
// the # of threads is > than the number of data elements!)
__global__ void primeP_gpu (unsigned int max, unsigned int *A, unsigned int *count)
{
 
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    printf("Thread: %d",n); 
    // do nothing if we are not in the useable space of
    // threads (see kernel launch call: you may be creating
    // more threads than you need)
    if (n >= max) return;
 
 
    atomicAdd(count, 1);
 
}
 
 
__global__ void single_image_global_gpu (unsigned float *image_vec, int tr, float *W, 
									float *word_vecs, 
									int word_vecs_count,
									float *Mv
									float *gradient){
	
	//doing everything by row
	int n=threadIdx.x;
	int dot_sum=0.0;
	for ( int i=0; i<4096; i++){
		int idx=n*4096 + i;
		dot_sum+=W[idx]*image_vec[i];
	}
	Mv[n]=dot_sum;
	
	__shared__ float label_word_vec[300];
	label_word_vec[n]=word_vecs[300*tr+n];
	
	
	__shared__ float w_label_Mv=0.0;
	atomicAdd(w_label_Mv,Mv[n]*label_word_vec[n]);

	__shared__ float sum_w_err[300];
	sum_w_err[n]=0.0;	
	
	__syncthreads();

	int n_loss=0;
	float loss=0.0;
		

	if(n==1){
		for(int i=0; i<word_vecs_count; i++){
			if(i!=tr){
				
				//calculate w_j_Mv
				int offset=i*300;
				float w_j_Mv=0.0;
				for(int j=0; j<300; j++)
					w_j_Mv+=Mv[j]+word_vecs[offset+j];

				float loss_j =  .1 - w_label_Mv + w_j_Mv
				if(loss_j>0){
                			n_loss    += 1
                			loss      += loss_j
			                for(int k=0;k<300;k++)
						sum_w_err[k] += word_vecs[offset+k];
					//breaking
					i=word_vecs_count;
				}	
			}
		}
	}

	
	
}

int main (int argc, char *argv[])
{

   //1. Need to get data and word2vec in correct format:
	//1-image vectors in 500,000 * 4096 float array
	//2-Corresponding image label
	//3-check if the label has a word vector, if not, throw out
	//(resultng in img_vecs (n*4096), img_labels (n,1), word_vecs (n,300)
	// n is the number of filtered image vectors
	
	// initialize weight matrix (4096*300)
	// put on global memory of the device

	//put word_vec matrix  (1000 * 300)
	//onto device global memory

	// for e in epochs:
		//for n in total_images/minibatch_size:
			
			//load all the image vectors (1 * 4096)* mini_batch size
			// or maximum amount of images

		//print out some validation if possible

	
	//Compute Gradient for a given matrix (single and minibatch):
		//have a previous weight matrix M (300*4096)
		
		//find Mv=M*img_vec (300*4096) dot (4096*1) = (300 * 1)
		
		//find word optimum: w_label_Mv (1 * 300) dot (300 *1) = scalar

		//for all labels:
			//find label within margin

				//find losss
		
		//derivative is outer product

		//step=gradient * step_rate * momentum

		//atomic_add step to weights



    //Simple error checking
    if(argc<3 || argc>4){
	printf("ERROR: Usage ./primeV filename number_of_integers number_of_threads(optional)\n");
	exit(EXIT_FAILURE);
    }
     
    printf("beginning\n");
 
    struct timeval t0, t1, t2;
 
    //Filename to read in:
    char* filename=argv[1];

    FILE* f=fopen(filename,"r");
    if( f == NULL ){
      perror("Error on file open.\n");
      exit(EXIT_FAILURE);
    }
 
    // How many integers are in the test file:
    unsigned int numIntegers = 1000000;
    if (sscanf(argv[2], "%i", &numIntegers)!=1){
	printf("Second argument must be the number of integers in file!\n");
    	exit(EXIT_FAILURE);
    }

    //Number of threads, defaults to 512 if not specified
    unsigned int numThreads=512;
    if(argc==4){
	if(sscanf(argv[3], "%i", &numThreads)!=1) {
		printf("Third argument must be number of threads per block\n");
		exit(EXIT_FAILURE);
	}
    }
 
    // start basic timing:
    gettimeofday (&t0, 0);
 
    // allocate the array to hold the data:
    unsigned int *h_intAArray;
    h_intAArray = (unsigned int *) malloc (numIntegers * sizeof (unsigned int));
 
    // read file for integers
    int number=0;
    for(int i=0;i<numIntegers; i++) {
	fscanf (f, "%d", &number); 
        h_intAArray[i] = number;
    }
     
    // count how many are prime:
    unsigned int primeCount = 0;
    for (int i = 0; i < numIntegers; ++i) {
        int isprime = primeP(h_intAArray, i);
        primeCount += isprime;
    }
 
    // how much time has elapsed?
    gettimeofday (&t1, 0);
 
    //
    // GPU version
    //
 
    // allocate the A array on the GPU, and copy the data over:
    unsigned int *d_intAArray;
 
    GPU_CHECKERROR(
    hipMalloc ((void **) &d_intAArray, numIntegers * sizeof (unsigned int))
    );
 
    GPU_CHECKERROR(
    hipMemcpy ((void *) d_intAArray,
                (void *) h_intAArray,
                numIntegers * sizeof (unsigned int),
                hipMemcpyHostToDevice)
    );
 
    // allocate a location to hold the count, and set it to zero:
    unsigned int *d_numprimes;
    hipMalloc ((void **) &d_numprimes, sizeof (unsigned int));
    hipMemset ((void *) d_numprimes, 0, sizeof (unsigned int));
 
 
    // we want to run a grid of 512-thread blocks (for reasons you
    // will understand later. How many such blocks will we need?
    // NOTE: be SURE to prevent integer division if you use this
    // snippet: that "1.0*" is absolutely required to prevent
    // rounding before the ceil() call:
    unsigned int threads_per_block;
    if (numThreads<1){
	threads_per_block = 512;
    } else {
	threads_per_block = numThreads;
    }
    unsigned int num_blocks = ceil (numIntegers / (1.0*threads_per_block) );
    printf("Using %d blocks, each with %d threads.\n",num_blocks, threads_per_block); 

    // launch the kernel:
    primeP_gpu<<<num_blocks, threads_per_block>>>
                                        (numIntegers,
                                        d_intAArray,
                                        d_numprimes);
 
    // get back the count:
    unsigned int h_numprimes;
 
    hipMemcpy ((void *) &h_numprimes,
                (void *) d_numprimes,
                sizeof(unsigned int),
                hipMemcpyDeviceToHost);
 
    // make sure the GPU is finished doing everything!
    hipDeviceSynchronize();
 
    // finish timing:
    gettimeofday (&t2, 0);
 
    // free up the memory:
    hipFree (d_intAArray);
    hipFree (d_numprimes);
    free (h_intAArray); 
 
    // complete the timing:
    float timdiff1 = (1000000.0*(t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)) / 1000000.0;
    float timdiff2 = (1000000.0*(t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec)) / 1000000.0;
 
    //printf ("done: time taken for serial version is %3.1f s\n", timdiff1);
    //printf ("done: time taken for parallel version is %3.1f s\n", timdiff2);
 
    //printf ("serial version found this many primes:%d \n", primeCount);
    //printf ("parallel version found this many primes:%d \n", h_numprimes);

    //print the result as specified
    printf("%d %3.1f %d %3.1f\n",primeCount, timdiff1, h_numprimes, timdiff2);
 
    printf("ending\n");
 
 
}
