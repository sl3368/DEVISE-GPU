#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdlib.h> 

// error checking for CUDA calls: use this around ALL your calls!
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

__global__ void single_image_global_gpu (float *image_vec, int *tr, float *W, 
                                    float *word_vecs, 
                                    int word_vecs_count,
                                    float *M_v,
                                    float *gradient,
                                    float momentum,
                                    float step_rate){

	//For each chunk
    for(int r=0; r<20; r++){
       
    	//doing everything by row
    	int block=blockIdx.x;
    	int n=threadIdx.x;

		//compute initial low-dimensional image vector Mv
    	int dot_sum=0.0;
    	__shared__ float Mv[300];

    	for ( int i=0; i<4096; i++){
    	    int idx=n*4096 + i;
    	    dot_sum+=W[idx]*image_vec[r*40960+block*4096+i];
    	}   

    	Mv[n]=dot_sum;
    	
		//initialize shared memory   
    	__shared__ float label_word_vec[300];
    	label_word_vec[n]=word_vecs[300*tr[r*10+block]+n];
    	
    	__shared__ float w_label_Mv[1];
    	w_label_Mv[0]=0.0;
    	atomicAdd(&w_label_Mv[0],Mv[n]*label_word_vec[n]);

    	__shared__ float sum_w_err[300];
    	sum_w_err[n]=0.0;   
    	   
    	__syncthreads();

    	int n_loss=0;
		float loss=0.0;

    	if(n==1){
    	    for(int i=0; i<word_vecs_count; i++){
    	        if(i!=tr[0]){
    	            //calculate w_j_Mv
    	            int offset=i*300;
    	            float w_j_Mv=0.0;
    	            for(int j=0; j<300; j++)
    	                w_j_Mv+=Mv[j]+word_vecs[offset+j];

					//calculate loss
    	            float loss_j = 0.1 - w_label_Mv[0] + w_j_Mv; //hard coding of margin
    	            if(loss_j>0){
    	                n_loss++;
    	                loss += loss_j;
    	                for(int k=0;k<300;k++)
    	                    sum_w_err[k] += word_vecs[offset+k];
    	                //breaking
    	                i=word_vecs_count;
    	            }
    	        }
    	    }
    	}

    	//scaling loss
    	float scale=0.0;
    	if (n_loss>0) scale=1000/n_loss;	//Hard coding right now, but should change

    	__syncthreads();

    	label_word_vec[n]+=sum_w_err[n]*n_loss; //label_word_vec is now error vector

    	//calculate outer product
    	//can move calculation and gradient step into the same loop
    	for(int j=0;j<4096; j++){
    	    atomicAdd(&W[n*4096+j],-1.0*scale*label_word_vec[n]*image_vec[j]*step_rate+momentum);
    	}

		__syncthreads();

    }
}

int main (int argc, char *argv[])
{

	// Number of input images
	int N = 20;
	// Number of validation images
    int M = N/4;   

	// Image vectors in N * 4096 float array
	float images[N*4096];
	float validation_images[M*4096];

	// Corresponding image labels
	int labels[N];
	int validation_labels[M];

	// Word vector array
	float host_word_vecs[1000*300];

	// Read input images from file
    FILE *fp;
    fp = fopen(argv[1],"r");

    char newline;

    for(int i=0;i<N;i++) {
        for(int j=0;j<4096;j++) {
            fscanf(fp, "%f%c", (images+4096*i+j), &newline);
        }
    }

    fclose(fp);

    // Read validation images from file
    fp = fopen(argv[2],"r");

    for(int i=0;i<M;i++) {
        for(int j=0;j<4096;j++) {
            fscanf(fp, "%f%c", (validation_images+4096*i+j),&newline);
        }
    }

    fclose(fp);

    //Read labels from file
    fp = fopen(argv[3],"r");
       
    for(int i=0;i<N;i++) {
            fscanf(fp, "%d%c", (labels+i), &newline);
    }   

    fclose(fp);
	
	//Read validation labels from file
    fp = fopen(argv[4],"r");

    for(int i=0;i<M;i++) {
            fscanf(fp, "%d%c", (validation_labels+i), &newline);
    }

    fclose(fp);

    // Read word vectors from file
    fp = fopen(argv[5],"r");

    for(int i=0;i<1000;i++) {
        for(int j=0;j<300;j++) {
            fscanf(fp, "%f%c", (host_word_vecs+300*i+j), &newline);
        }
    }

    fclose(fp);

	// create timers
	hipEvent_t     start, stop;
    float           elapsedTime;

    // start the timers
    GPU_CHECKERROR( hipEventCreate( &start ) );
    GPU_CHECKERROR( hipEventCreate( &stop ) );

	// initialize host weight matrix (4096*300)
	float host_W[4096*300];

	// initialize weight matrix (4096*300)
	float *W;
	GPU_CHECKERROR(
		hipMalloc((void**) &W, 4096*300*sizeof(float))
	);
	hipMemset ((void *) W, 0, 4096*300*sizeof (unsigned int));

	//put word_vec matrix  (1000 * 300)
	//onto device global memory
	float *word_vecs;
	GPU_CHECKERROR(
		hipMalloc((void**) &word_vecs, 1000 * 300 * sizeof(float))
	);
	GPU_CHECKERROR(
    		hipMemcpy ((void *) word_vecs,
                (void *) host_word_vecs,
                1000 * 300 * sizeof (unsigned int),
                hipMemcpyHostToDevice)
   	); 

	int minibatch_size=20*10;

	// Container for minibatch of images on device
	float *image_vecs; 	
	GPU_CHECKERROR(
		hipMalloc((void**) &image_vecs, minibatch_size * 4096 * sizeof(float))
	);

	// True labels for the minibatch of images
	int *tr;
	GPU_CHECKERROR(
		hipMalloc((void**) &tr, minibatch_size * sizeof(int))
	);

	// Gradients of minibatch of images
	float *gradients;
	GPU_CHECKERROR(
		hipMalloc((void**) &gradients, minibatch_size * 300 * 4096 * sizeof(int))
	);

	// Low dimensional image vector Mv
	float *Mv;
	GPU_CHECKERROR(
		hipMalloc((void**) &Mv, 300 * sizeof(float))
	);

	hipStream_t    stream0, stream1;
	GPU_CHECKERROR( hipStreamCreate( &stream0 ) );	
	GPU_CHECKERROR( hipStreamCreate( &stream1 ) );	
	int num_epochs=1;

	GPU_CHECKERROR( hipEventRecord( start, 0 ) );
	
	//For ith epoch (i.e. ith run over data )
	for(int i=0;i<num_epochs;i++) {
		//For jth image 
		for(int j=0;j<N;j+= minibatch_size*2) {
				//create chunk for images and labels
				float *img_vec_chunk_0=images+(4096*j);							//image chunk for stream0
				float *img_vec_chunk_1=images+(4096*j)+minibatch_size*4096;		//image chunk for stream1
				int *img_labels_chunk_0=labels+j;								//label chunk for stream0	
				int *img_labels_chunk_1=labels+j+minibatch_size;				//label chunk for stream1	
				
				//first stream of image and vector chunks to GPU
			    GPU_CHECKERROR ( hipMemcpyAsync ((void *) image_vecs, (void *) img_vec_chunk_0,					
										minibatch_size* 4096 * sizeof (float),
    			            			hipMemcpyHostToDevice,
										stream0) );
				
			    GPU_CHECKERROR ( hipMemcpyAsync ( (void *) tr, (void *) img_labels_chunk_0,
										minibatch_size * sizeof (int),
    			            			hipMemcpyHostToDevice,
										stream0) );
				
				//run kernel
				single_image_global_gpu<<<10, 300, 0, stream0>>>
                                        (image_vecs,							//image vectors on GPU
                                        tr,										//true labels 
                                        W,										//weight matrix
										word_vecs,								//word vectors for all 1000 classes
										1000,									//number of classes
										Mv,										//low dimensional image vector
										gradients,								//gradients of mini-batch of images
										.9,										//momentum
										.0001);									//step_rate

				//second stream of image and vector chunks to GPU
			    GPU_CHECKERROR ( hipMemcpyAsync ((void *) image_vecs, (void *) img_vec_chunk_1,					
										minibatch_size* 4096 * sizeof (float),
    			            			hipMemcpyHostToDevice,
										stream1) );
				
		    	GPU_CHECKERROR ( hipMemcpyAsync ( (void *) tr, (void *) img_labels_chunk_1,
										minibatch_size * sizeof (int),
    			            			hipMemcpyHostToDevice,
										stream1) );
				
				//run kernel
				single_image_global_gpu<<<10, 300, 0, stream1>>>
                                        (image_vecs,							//image vectors on GPU
                                        tr,										//true labels 
                                        W,										//weight matrix
										word_vecs,								//word vectors for all 1000 classes
										1000,									//number of classes
										Mv,										//low dimensional image vector
										gradients,								//gradients of mini-batch of images
										.9,										//momentum
										.0001);									//step_rate

		}

		// Pull out weights after each epoch and calculate validation accuracy
        GPU_CHECKERROR ( hipMemcpyAsync ( (void *) &host_W, (void *) W,
                            4096*300* sizeof (float),
                            hipMemcpyHostToDevice,
                            stream1) );

        // Calculate validation accuracy here
 
	}

	GPU_CHECKERROR( hipStreamSynchronize( stream0 ) );
	GPU_CHECKERROR( hipStreamSynchronize( stream1 ) );

	//Time the kernel run
	GPU_CHECKERROR( hipEventRecord( stop, 0 ) );

    GPU_CHECKERROR( hipEventSynchronize( stop ) );
    GPU_CHECKERROR( hipEventElapsedTime( &elapsedTime,
                start, stop ) );

    printf( "Time taken:  %3.1f ms\n", elapsedTime );

	//Free device memory
	GPU_CHECKERROR( hipFree( W ) );
	GPU_CHECKERROR( hipFree( word_vecs ) );
	GPU_CHECKERROR( hipFree( image_vecs ) );
	GPU_CHECKERROR( hipFree( tr ) );
	GPU_CHECKERROR( hipFree( gradients ) );
	GPU_CHECKERROR( hipFree( Mv ) );

	//Destroy streams
	GPU_CHECKERROR( hipStreamDestroy( stream0 ) );
	GPU_CHECKERROR( hipStreamDestroy( stream1 ) );

 
}
